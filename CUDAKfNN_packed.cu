#include "hip/hip_runtime.h"
/*******************************************************************
*   CUDAKfNN_packed.cu
*   CUDAKfNN_packed
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Oct 21, 2016
*******************************************************************/
//
// Fastest GPU implementation of a brute-force
// matcher for 128-float descriptors such as SIFT
// in 2NN mode, i.e., a match is returned if the best
// match between a query vector and a training vector
// is more than a certain threshold ratio
// better than the second-best match.
//
// Float descriptors are slow. Check out my CUDAK2NN project
// for much faster binary description matching. Use a
// good binary descriptor such as LATCH where possible.
//
// That said, this laboriously crafted kernel is EXTREMELY fast
// for a float matcher.
//
// CUDA CC 3.0 or higher is required.
//
// All functionality is contained in the files CUDAKfNN_packed.h
// and CUDAKfNN_packed.cu. 'main.cpp' is simply a sample test harness
// with example usage and performance testing.
//

#include "CUDAKfNN_packed.h"

// 1.603

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDAKfNN_packed_kernel(const hipTextureObject_t tex_q, const int num_q, const hipTextureObject_t tex_t, const int num_t, int* const __restrict__ g_match, const float threshold) {
	int ofs_t = threadIdx.x & 7;
	uint4 train = tex1Dfetch<uint4>(tex_t, ofs_t);
	ofs_t += 8;
	uint4 q[2];
	for (int i = 0, ofs_q = ((threadIdx.x & 24) << 1) + (threadIdx.x & 7) + (blockIdx.x << 9) + (threadIdx.y << 6); i < 2; ++i, ofs_q += 8) q[i] = tex1Dfetch<uint4>(tex_q, ofs_q);
	int best_i;
	uint32_t best_v = 1000000000, second_v = 2000000000;
#pragma unroll 7
	for (int t = 0; t < num_t; ++t, ofs_t += 8) {
		uint32_t dist[2];
#pragma unroll
		for (int i = 0; i < 2; ++i) {
			uint32_t diffs = __vabsdiffu4(q[i].w, train.w);
			uint32_t tmp = __byte_perm(0U, diffs, 4U);
			dist[i] = tmp * tmp;
			tmp = __byte_perm(0U, diffs, 5U);
			dist[i] += tmp * tmp;
			tmp = (diffs >> 16) & 0xFF;
			dist[i] += tmp * tmp;
			tmp = (diffs >> 24) & 0xFF;
			dist[i] += tmp * tmp;

			diffs = __vabsdiffu4(q[i].x, train.x);
			tmp = __byte_perm(0U, diffs, 4U);
			dist[i] += tmp * tmp;
			tmp = __byte_perm(0U, diffs, 5U);
			dist[i] += tmp * tmp;
			tmp = (diffs >> 16) & 0xFF;
			dist[i] += tmp * tmp;
			tmp = (diffs >> 24) & 0xFF;
			dist[i] += tmp * tmp;

			diffs = __vabsdiffu4(q[i].y, train.y);
			tmp = __byte_perm(0U, diffs, 4U);
			dist[i] += tmp * tmp;
			tmp = __byte_perm(0U, diffs, 5U);
			dist[i] += tmp * tmp;
			tmp = (diffs >> 16) & 0xFF;
			dist[i] += tmp * tmp;
			tmp = (diffs >> 24) & 0xFF;
			dist[i] += tmp * tmp;

			diffs = __vabsdiffu4(q[i].z, train.z);
			tmp = __byte_perm(0U, diffs, 4U);
			dist[i] += tmp * tmp;
			tmp = __byte_perm(0U, diffs, 5U);
			dist[i] += tmp * tmp;
			tmp = (diffs >> 16) & 0xFF;
			dist[i] += tmp * tmp;
			tmp = (diffs >> 24) & 0xFF;
			dist[i] += tmp * tmp;
		}
		for (int i = 0; i < 2; ++i) dist[i] += __shfl_xor(dist[i], 1);
		train = tex1Dfetch<uint4>(tex_t, ofs_t);
		if (threadIdx.x & 1) dist[0] = dist[1];
		dist[0] += __shfl_xor(dist[0], 2);
		second_v = min(dist[0] += __shfl_xor(dist[0], 4), second_v);
		if (dist[0] < best_v) {
			second_v = best_v;
			best_i = t;
			best_v = dist[0];
		}
	}
	const int idx = (blockIdx.x << 6) + (threadIdx.y << 3) + ((threadIdx.x & 24) >> 2) + (threadIdx.x & 7);
	if (idx < num_q && ((threadIdx.x & 6) == 0)) g_match[idx] = static_cast<float>(best_v) > threshold * static_cast<float>(second_v) ? -1 : best_i;
}

void CUDAKfNN_packed(const hipTextureObject_t tex_t, const int num_t, const hipTextureObject_t tex_q, const int num_q, int* const __restrict d_m, const float threshold) {
	CUDAKfNN_packed_kernel<<<((num_q - 1) >> 6) + 1, { 32, 8 }>>>(tex_q, num_q, tex_t, num_t, d_m, threshold*threshold);
	hipDeviceSynchronize();
}